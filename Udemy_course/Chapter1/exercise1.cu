#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void print_details()
{
 printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, threadIdx.x : %d,threadIdx.y : %d, threadIdx.z : %d,gridDim.x : %d, gridDim.y : %d, gridDim.z : %d \n", blockIdx.x, blockIdx.y, blockIdx.z, 
	 threadIdx.x, threadIdx.y, threadIdx.z, gridDim.x, gridDim.y, gridDim.z); 

}

int main()
{
int nx = 4; 
int ny = 4; 
int nz = 4; 
dim3 block(2,2,2); 
dim3 grid(nx/block.x, ny/block.y, nz/block.z); 

print_details<<<grid,block>>>(); 

hipDeviceSynchronize(); 
hipDeviceReset(); 
return 0;
}
