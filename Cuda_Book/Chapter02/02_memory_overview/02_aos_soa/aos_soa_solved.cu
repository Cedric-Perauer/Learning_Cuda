
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<stdlib.h>
#include<string.h>


#define NUM_THREADS 256

#define IMG_SIZE 1048576

struct Coefficients_AOS {
  int* r;
  int* b;
  int* g;
  int* hue;
  int* saturation;
  int* maxVal;
  int* minVal;
  int* finalVal; 
};


__global__
void complicatedCalculation(Coefficients_AOS  data)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;


  int grayscale = (data.r[i] + data.g[i] + data.b[i])/data.maxVal[i];
  int hue_sat = data.hue[i] * data.saturation[i] / data.minVal[i];
  data.finalVal[i] = grayscale*hue_sat; 
}

void complicatedCalculation()
{

  Coefficients_AOS d_x;

  hipMalloc(&d_x.r, IMG_SIZE*sizeof(int)); 
  hipMalloc(&d_x.g, IMG_SIZE*sizeof(int)); 
  hipMalloc(&d_x.b, IMG_SIZE*sizeof(int)); 
  hipMalloc(&d_x.hue, IMG_SIZE*sizeof(int)); 
  hipMalloc(&d_x.saturation, IMG_SIZE*sizeof(int)); 
  hipMalloc(&d_x.maxVal, IMG_SIZE*sizeof(int)); 
  hipMalloc(&d_x.minVal, IMG_SIZE*sizeof(int)); 
  hipMalloc(&d_x.finalVal, IMG_SIZE*sizeof(int)); 


  int num_blocks = IMG_SIZE/NUM_THREADS;

  complicatedCalculation<<<num_blocks,NUM_THREADS>>>(d_x);

  
  hipFree(d_x.r);
  hipFree(d_x.g);
  hipFree(d_x.b);
  hipFree(d_x.hue);
  hipFree(d_x.saturation);
  hipFree(d_x.maxVal);
  hipFree(d_x.maxVal);
  hipFree(d_x.minVal);
  hipFree(d_x.finalVal);
}



int main(int argc, char*argv[])
{

	complicatedCalculation();
	return 0;
}






