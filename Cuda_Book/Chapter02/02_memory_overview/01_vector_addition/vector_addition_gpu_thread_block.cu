
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 512

void host_add(int *a, int *b, int *c) {
	for(int idx=0;idx<N;idx++)
		c[idx] = a[idx] + b[idx];
}

__global__ void device_add(int *a, int *b, int *c) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
        c[index] = a[index] + b[index];
}


//basically just fills the array with index.
void fill_array(int *data) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int*c) {
	for(int idx=0;idx<N;idx++)
		printf("\n %d + %d  = %d",  a[idx] , b[idx], c[idx]);
}
int main(void) {
	int *a, *b, *c;
        int *d_a, *d_b, *d_c; // device copies of a, b, c
	int threads_per_block=0, no_of_blocks=0;

	int size = N * sizeof(int);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); fill_array(a);
	b = (int *)malloc(size); fill_array(b);
	c = (int *)malloc(size);

        // Alloc space for device copies of a, b, c
        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);

       // Copy inputs to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	threads_per_block = 4;
	no_of_blocks = N/threads_per_block;	
	device_add<<<no_of_blocks,threads_per_block>>>(d_a,d_b,d_c);

        // Copy result back to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	print_output(a,b,c);

	free(a); free(b); free(c);
        hipFree(d_a); hipFree(d_b); hipFree(d_c);



	return 0;
}
