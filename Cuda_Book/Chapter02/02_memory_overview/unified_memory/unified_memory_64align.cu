
#include <hip/hip_runtime.h>
#include <iostream>
#include<stdio.h>
#include <math.h>

#define STRIDE_64K 65536

__global__ void init(int n, float *x, float *y) {

  int lane_id = threadIdx.x & 31;
  size_t warp_id = (threadIdx.x + blockIdx.x * blockDim.x) >> 5;
  size_t warps_per_grid = (blockDim.x * gridDim.x) >> 5;
  size_t warp_total = ((sizeof(float)*n) + STRIDE_64K-1) / STRIDE_64K;


if(blockIdx.x==0 && threadIdx.x==0) {
	//printf("\n TId[%d] ", threadIdx.x);
	//printf(" WId[%u] ", warp_id);
	//printf(" LId[%u] ", lane_id);
	//printf(" WperG[%u] ", warps_per_grid);
	//printf(" wTot[%u] ", warp_total);
	//printf(" rep[%d] ", STRIDE_64K/sizeof(float)/32);
}
  for(; warp_id < warp_total; warp_id += warps_per_grid) {
    #pragma unroll
    for(int rep = 0; rep < STRIDE_64K/sizeof(float)/32; rep++) {
      size_t ind = warp_id * STRIDE_64K/sizeof(float) + rep * 32 + lane_id;
      if (ind < n) {
        x[ind] = 1.0f;
//if(blockIdx.x==0 && threadIdx.x==0) {
//	printf(" \nind[%d] ", ind);
//} 
        y[ind] = 2.0f;
      }
    }
  }

}
 
// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}
 
int main(void)
{
  int N = 1<<20;
  float *x, *y;
 
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
 
 
  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  size_t warp_total = ((sizeof(float)*N) + STRIDE_64K-1) / STRIDE_64K;
  int numBlocksInit = (warp_total*32) / blockSize;
  
  init<<<numBlocksInit, blockSize>>>(N, x, y);
  add<<<numBlocks, blockSize>>>(N, x, y);
 
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
 
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
 
  // Free memory
  hipFree(x);
  hipFree(y);
 
  return 0;
}
