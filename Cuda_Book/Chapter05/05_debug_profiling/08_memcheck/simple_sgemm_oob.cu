#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"

#define BLOCK_DIM 16

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on GPU
//! C = alpha * A * B + beta * C
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param C          matrix C as provided to device
//! @param N          height of matrix A and matrix C
//! @param M          width of matrix B and matrix C
//! @param K          width of matrix A and height of matrix C
//! @param alpha      scala value for matrix multiplication
//! @param beta       scala value for matrix summation with C
////////////////////////////////////////////////////////////////////////////////
__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    row += 1;

    float sum = 0.f;
    for (int i = 0; i < K; ++i) 
        sum += A[row * K + i] * B[i * K + col];

    C[row * M + col] = alpha * sum + beta * C[row * M + col];
}

void random_init(float *data, int size)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }
}

int main()
{
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    int N, M, K;
    float alpha = 2.f;
    float beta = 1.f;
    int n_iter = 1;
    N = M = K = 2048;

    // allocation of linear memory space
    A = (float *)malloc(N * K * sizeof(float));
    B = (float *)malloc(K * M * sizeof(float));
    C = (float *)malloc(N * M * sizeof(float));

    // allocation of gpu linear memory space
    checkCudaErrors(hipMalloc((void **)&d_A, N * K * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_B, K * M * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_C, N * M * sizeof(float)));

    // initialize randomized values for memory space
    random_init(A, N * K);
    random_init(B, K * M);
    random_init(C, N * M);

    // copy initial value for gpu memory
    checkCudaErrors(hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, B, K * M * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C, C, N * M * sizeof(float), hipMemcpyHostToDevice));

    // do operation
    for (int i = 0; i < n_iter; i++) {
        dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
        dim3 dimGrid(M / dimBlock.x, N / dimBlock.y);
        sgemm_kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N, M, K, alpha, beta);
        checkCudaErrors(hipGetLastError());
    }

    checkCudaErrors(hipDeviceSynchronize());
    printf("Application finished successfully.");

    // terminates allocated gpu memory space
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    // terminates allocated memory space
    free(A);
    free(B);
    free(C);

    return 0;
}
