#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"

#define BLOCK_DIM 16

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on GPU
//! C = alpha * A * B + beta * C
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param C          matrix C as provided to device
//! @param N          height of matrix A and matrix C
//! @param M          width of matrix B and matrix C
//! @param K          width of matrix A and height of matrix C
//! @param alpha      scala value for matrix multiplication
//! @param beta       scala value for matrix summation with C
////////////////////////////////////////////////////////////////////////////////
__global__ void sgemm_kernel_A(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;
    for (int i = 0; i < K; ++i) 
        sum += A[row * K + i] * B[i * K + col];

    C[row * M + col] = alpha * sum + beta * C[row * M + col];
}

void sgemm_gpu_A(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid(M / dimBlock.x, N / dimBlock.y);
    sgemm_kernel_A<<<dimGrid, dimBlock>>>(A, B, C, N, M, K, alpha, beta);
    checkCudaErrors(hipGetLastError());
}

__global__ void sgemm_kernel_B(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;
    for (int i = 0; i < K; ++i)
        sum += A[row * K + i] * B[i * K + col];

    C[row * M + col] = alpha * sum + beta * C[row * M + col];
}

void sgemm_gpu_B(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid(M / dimBlock.x, N / dimBlock.y);
    sgemm_kernel_B<<<dimGrid, dimBlock>>>(A, B, C, N, M, K, alpha, beta);
    checkCudaErrors(hipGetLastError());
}

void random_init(float *data, int size)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }
}

int main()
{
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    int N, M, K;
    float alpha = 2.f;
    float beta = 1.f;
    int n_iter = 5;
    N = M = K = 2048;

    // allocation of linear memory space
    A = (float *)malloc(N * K * sizeof(float));
    B = (float *)malloc(K * M * sizeof(float));
    C = (float *)malloc(N * M * sizeof(float));

    // allocation of gpu linear memory space
    checkCudaErrors(hipMalloc((void **)&d_A, N * M * sizeof(float))); // FIX ME (1)
    checkCudaErrors(hipMalloc((void **)&d_B, K * M * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_C, N * M * sizeof(float)));

    // initialize randomized values for memory space
    random_init(A, N * K);
    random_init(B, K * M);
    random_init(C, N * M);

    // copy initial value for gpu memory
    checkCudaErrors(hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice)); 
    checkCudaErrors(hipMemcpy(d_B, A, K * M * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_C, A, N * M * sizeof(float), hipMemcpyHostToDevice));

    // do operation
    for (int i = 0; i < n_iter; i++)
        sgemm_gpu_A(d_A, d_B, NULL, N, M, K, alpha, beta); // FIX ME (2)

    for (int i = 0; i < n_iter; i++)
        sgemm_gpu_B(d_A, d_B, d_C, N, M, K, alpha, beta);

    checkCudaErrors(hipDeviceSynchronize());
    printf("Application finished successfully.");

    // terminates allocated gpu memory space
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    // terminates allocated memory space
    free(A);
    free(B);
    free(C);

    return 0;
}
