
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std;

__global__ void
simple_saxpy_kernel(float *y, const float* x, const float alpha, const float beta)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    y[idx] = alpha * x[idx] + beta;
}

__global__ void
iterative_saxpy_kernel(float *y, const float* x, const float alpha, const float beta, int n_loop)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < n_loop; i++)
        y[idx] = alpha * x[idx] + beta;
}

__global__ void
recursive_saxpy_kernel(float *y, const float* x, const float alpha, const float beta, int depth)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (depth == 0)
        return;
    else 
        y[idx] = alpha * x[idx] + beta;
    
    if (threadIdx.x == 0) 
        recursive_saxpy_kernel<<< 1, blockDim.x >>>(y, x, alpha, beta, depth - 1);
}

int main()
{
    float *d_y, *d_x;
    int size = 1 << 10;
    int bufsize = size * sizeof(float);
    int n_loop = 24;
    float elapsed_time_A, elapsed_time_B, elapsed_time_C;
    float alpha = 0.1f, beta = 0.2f;

    hipEvent_t start, stop;

    // initialize cuda event
    hipEventCreateWithFlags(&start, hipEventBlockingSync);
    hipEventCreateWithFlags(&stop, hipEventBlockingSync);

    hipMalloc((void**)&d_y, bufsize);
    hipMalloc((void**)&d_x, bufsize);

    int dimBlock = 256;
    int dimGrid = size / dimBlock;

    // Step 1. Loop outside the kernel
    hipEventRecord(start);
    for (int i = 0; i < n_loop; i++) {
        simple_saxpy_kernel<<< dimGrid, dimBlock >>>(d_y, d_x, alpha, beta);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_A, start, stop);

    // Step 2. Loop inside the kernel
    hipEventRecord(start);
    iterative_saxpy_kernel<<< dimGrid, dimBlock >>>(d_y, d_x, alpha, beta, n_loop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_B, start, stop);

    // Step 3. Loop with the recursion
    hipEventRecord(start);
    recursive_saxpy_kernel<<< dimGrid, dimBlock >>>(d_y, d_x, alpha, beta, n_loop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_C, start, stop);

    printf("Elapsed Time...\n");
    printf("simple loop: %.3f ms\n", elapsed_time_A);
    printf("inner loop : %.3f ms\n", elapsed_time_B);
    printf("recursion  : %.3f ms\n", elapsed_time_C);

    hipFree(d_y);
    hipFree(d_x);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}