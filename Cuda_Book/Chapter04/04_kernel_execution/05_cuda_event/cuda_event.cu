#include "hip/hip_runtime.h"
#include <cstdio>
#include <helper_timer.h>

using namespace std;

__global__ void vecAdd_kernel(float *c, const float* a, const float* b);
void init_buffer(float *data, const int size);

int main(int argc, char* argv[])
{
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 24;
    int bufsize = size * sizeof(float);

    // allocate host memories
    hipHostMalloc((void**)&h_a, bufsize);
    hipHostMalloc((void**)&h_b, bufsize);
    hipHostMalloc((void**)&h_c, bufsize);

    // initialize host values
    srand(2019);
    init_buffer(h_a, size);
    init_buffer(h_b, size);
    init_buffer(h_c, size);

    // allocate device memories
    hipMalloc((void**)&d_a, bufsize);
    hipMalloc((void**)&d_b, bufsize);
    hipMalloc((void**)&d_c, bufsize);

    // copy host -> device
    hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice);

    // initialize the host timer
    StopWatchInterface *timer;
    sdkCreateTimer(&timer);

    hipEvent_t start, stop;
    // create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to measure the execution time
    sdkStartTimer(&timer);
    hipEventRecord(start);

    // launch cuda kernel
    dim3 dimBlock(256);
    dim3 dimGrid(size / dimBlock.x);
    vecAdd_kernel<<< dimGrid, dimBlock >>>(d_c, d_a, d_b);

    // record the event right after the kernel execution finished
    hipEventRecord(stop);

    // Synchronize the device to measure the execution time from the host side
    hipEventSynchronize(stop); // we also can make synchronization based on CUDA event
    sdkStopTimer(&timer);
    
    // copy device -> host
    hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost);

    // print out the result
    int print_idx = 256;
    printf("compared a sample result...\n");
    printf("host: %.6f, device: %.6f\n",  h_a[print_idx] + h_b[print_idx], h_c[print_idx]);

    // print estimated kernel execution time
    float elapsed_time_msed = 0.f;
    hipEventElapsedTime(&elapsed_time_msed, start, stop);
    printf("CUDA event estimated - elapsed %.3f ms \n", elapsed_time_msed);

    // Compute and print the performance
    elapsed_time_msed = sdkGetTimerValue(&timer);
    printf("Host measured time= %.3f msec/s\n", elapsed_time_msed);

    // terminate device memories
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // terminate host memories
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    // delete timer
    sdkDeleteTimer(&timer);

    // terminate CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}

__global__ void
vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < 500; i++)
        c[idx] = a[idx] + b[idx];
}

void init_buffer(float *data, const int size)
{
    for (int i = 0; i < size; i++) 
        data[i] = rand() / (float)RAND_MAX;
}