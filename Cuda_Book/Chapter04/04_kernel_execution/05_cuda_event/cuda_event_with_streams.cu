#include "hip/hip_runtime.h"
#include <cstdio>
#include <helper_timer.h>

using namespace std;

__global__ void vecAdd_kernel(float *c, const float* a, const float* b);
void init_buffer(float *data, const int size);

class Operator
{
private:
    int index;
    StopWatchInterface *p_timer;

    static void CUDART_CB Callback(hipStream_t stream, hipError_t status, void* userData);
    void print_time();

    hipEvent_t start, stop;

protected:
    hipStream_t stream = nullptr;

public:
    Operator(bool create_stream = true) {
        if (create_stream)
            hipStreamCreate(&stream);
        sdkCreateTimer(&p_timer);

        // create CUDA events
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~Operator() {
        if (stream != nullptr)
            hipStreamDestroy(stream);
        sdkDeleteTimer(&p_timer);

        // terminate CUDA events
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void set_index(int idx) { index = idx; }
    void async_operation(float *h_c, const float *h_a, const float *h_b,
                          float *d_c, float *d_a, float *d_b,
                          const int size, const int bufsize);
    void print_kernel_time();

}; // Operator

void Operator::CUDART_CB Callback(hipStream_t stream, hipError_t status, void* userData) {
    Operator* this_ = (Operator*) userData;
    this_->print_time();
}

void Operator::print_time() {
    // end timer
    sdkStopTimer(&p_timer);
    float elapsed_time_msed = sdkGetTimerValue(&p_timer);
    printf("stream %2d - elapsed %.3f ms \n", index, elapsed_time_msed);
}

void Operator::print_kernel_time() {
    float elapsed_time_msed = 0.f;
    hipEventElapsedTime(&elapsed_time_msed, start, stop);
    printf("kernel in stream %2d - elapsed %.3f ms \n", index, elapsed_time_msed);
}

void Operator::async_operation(float *h_c, const float *h_a, const float *h_b,
                          float *d_c, float *d_a, float *d_b,
                          const int size, const int bufsize)
{
    // start timer
    sdkStartTimer(&p_timer);

    // copy host -> device
    hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice, stream);

    // record the event before the kernel execution
    hipEventRecord(start, stream);

    // launch cuda kernel
    dim3 dimBlock(256);
    dim3 dimGrid(size / dimBlock.x);
    vecAdd_kernel<<< dimGrid, dimBlock, 0, stream >>>(d_c, d_a, d_b);

    // record the event right after the kernel execution finished
    hipEventRecord(stop, stream);

    // copy device -> host
    hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost, stream);

    // what happen if we include CUDA event synchronize?
    // QUIZ: hipEventSynchronize(stop);

    // register callback function
    hipStreamAddCallback(stream, Operator::Callback, this, 0);
}

class Operator_with_priority: public Operator {
public:
    Operator_with_priority() : Operator(false) {}

    void set_priority(int priority) {
        hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, priority);
    }
};

int main(int argc, char* argv[])
{
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 24;
    int bufsize = size * sizeof(float);
    int num_operator = 4;

    if (argc != 1)
        num_operator = atoi(argv[1]);

    // check the current device supports CUDA stream's prority
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); 
    if (prop.streamPrioritiesSupported == 0) {
        printf("This device does not support priority streams");
        return 1;
    }

    // initialize timer
    StopWatchInterface *timer;
    sdkCreateTimer(&timer);

    // allocate host memories
    hipHostMalloc((void**)&h_a, bufsize);
    hipHostMalloc((void**)&h_b, bufsize);
    hipHostMalloc((void**)&h_c, bufsize);

    // initialize host values
    srand(2019);
    init_buffer(h_a, size);
    init_buffer(h_b, size);
    init_buffer(h_c, size);

    // allocate device memories
    hipMalloc((void**)&d_a, bufsize);
    hipMalloc((void**)&d_b, bufsize);
    hipMalloc((void**)&d_c, bufsize);

    // create list of operation elements
    Operator_with_priority *ls_operator = new Operator_with_priority[num_operator];

    // Get Priority range
    int priority_low, priority_high;
    hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);
    printf("Priority Range: low(%d), high(%d)\n", priority_low, priority_high);

    // start to measure the execution time
    sdkStartTimer(&timer);
    
    // execute each operator collesponding data
    // priority setting for each CUDA stream
    for (int i = 0; i < num_operator; i++) {
        // int offset = i * size / num_operator;
        ls_operator[i].set_index(i);
        if (i + 1 == num_operator)
            ls_operator[i].set_priority(priority_high);
        else
            ls_operator[i].set_priority(priority_low);
    }

    // operation (copy(H2D), kernel execution, copy(D2H))
    for (int i = 0; i < num_operator; i++) {
        int offset = i * size / num_operator;
        ls_operator[i].async_operation(&h_c[offset], &h_a[offset], &h_b[offset],
                                       &d_c[offset], &d_a[offset], &d_b[offset],
                                       size / num_operator, bufsize / num_operator);
    }

    // synchronize all the stream operation
    hipDeviceSynchronize();

    // stop to measure the execution time    
    sdkStopTimer(&timer);

    // print each cuda stream execution time
    for (int i = 0; i < num_operator; i++)
        ls_operator[i].print_kernel_time(); 

    // print out the result
    int print_idx = 256;
    printf("compared a sample result...\n");
    printf("host: %.6f, device: %.6f\n",  h_a[print_idx] + h_b[print_idx], h_c[print_idx]);

    // Compute and print the performance
    float elapsed_time_msed = sdkGetTimerValue(&timer);
    float bandwidth = 3 * bufsize * sizeof(float) / elapsed_time_msed / 1e6;
    printf("Time= %.3f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    // delete timer
    sdkDeleteTimer(&timer);

    // terminate operators
    delete [] ls_operator;

    // terminate device memories
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // terminate host memories
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    
    return 0;
}

__global__ void
vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < 500; i++)
        c[idx] = a[idx] + b[idx];
}

void init_buffer(float *data, const int size)
{
    for (int i = 0; i < size; i++) 
        data[i] = rand() / (float)RAND_MAX;
}