#include "hip/hip_runtime.h"
#include <cstdio>
#include <helper_timer.h>

using namespace std;

__global__ void vecAdd_kernel(float *c, const float* a, const float* b);
void init_buffer(float *data, const int size);

class Operator
{
private:
    int index;
    hipStream_t stream;

public:
    Operator() {
        hipStreamCreate(&stream);
    }

    ~Operator() {
        hipStreamDestroy(stream);
    }

    void set_index(int idx) { index = idx; }
    void async_operation(float *h_c, const float *h_a, const float *h_b,
                          float *d_c, float *d_a, float *d_b,
                          const int size, const int bufsize);

}; // Operator

void Operator::async_operation(float *h_c, const float *h_a, const float *h_b,
                          float *d_c, float *d_a, float *d_b,
                          const int size, const int bufsize)
{
    // copy host -> device
    hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice, stream);

    // launch cuda kernel
    dim3 dimBlock(256);
    dim3 dimGrid(size / dimBlock.x);
    vecAdd_kernel<<< dimGrid, dimBlock, 0, stream >>>(d_c, d_a, d_b);

    // copy device -> host
    hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    printf("Launched GPU task %d\n", index);
}

int main(int argc, char* argv[])
{
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 24;
    int bufsize = size * sizeof(float);
    int num_operator = 4;

    if (argc != 1)
        num_operator = atoi(argv[1]);
    
    // allocate host memories
    hipHostMalloc((void**)&h_a, bufsize);
    hipHostMalloc((void**)&h_b, bufsize);
    hipHostMalloc((void**)&h_c, bufsize);

    // initialize host values
    srand(2019);
    init_buffer(h_a, size);
    init_buffer(h_b, size);
    init_buffer(h_c, size);

    // allocate device memories
    hipMalloc((void**)&d_a, bufsize);
    hipMalloc((void**)&d_b, bufsize);
    hipMalloc((void**)&d_c, bufsize);

    // create list of operation elements
    Operator *ls_operator = new Operator[num_operator];

    // initialize & start timer
    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // execute each operator collesponding data
    for (int i = 0; i < num_operator; i++) {
        int offset = i * size / num_operator;
        ls_operator[i].set_index(i);
        ls_operator[i].async_operation(&h_c[offset], &h_a[offset], &h_b[offset],
                                       &d_c[offset], &d_a[offset], &d_b[offset],
                                       size / num_operator, bufsize / num_operator);
    }

    // synchronize until all the stream operation is finished
    hipDeviceSynchronize();

    sdkStopTimer(&timer);

    // print out the result
    int print_idx = 256;
    printf("compared a sample result...\n");
    printf("host: %.6f, device: %.6f\n",  h_a[print_idx] + h_b[print_idx], h_c[print_idx]);

    // Compute and print the performance
    float elapsed_time_msed = sdkGetTimerValue(&timer);
    float bandwidth = 3 * bufsize * sizeof(float) / elapsed_time_msed / 1e6;
    printf("Time= %.3f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

    // terminate operators
    delete [] ls_operator;

    // terminate device memories
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // terminate host memories
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    
    return 0;
}

void init_buffer(float *data, const int size)
{
    for (int i = 0; i < size; i++) 
        data[i] = rand() / (float)RAND_MAX;
}

__global__ void
vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < 500; i++)
        c[idx] = a[idx] + b[idx];
}