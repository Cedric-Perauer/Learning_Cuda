
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std;

__global__ void
foo_kernel(int step)
{
    printf("loop: %d\n", step);
}

int main()
{
    int n_stream = 5;
    hipStream_t *ls_stream;
    ls_stream = (hipStream_t*) new hipStream_t[n_stream];

    // create multiple streams
    for (int i = 0; i < n_stream; i++)
        hipStreamCreate(&ls_stream[i]);

    // execute kernels with the CUDA stream each
    for (int i = 0; i < n_stream; i++)
        if (i == 3)
            foo_kernel<<< 1, 1, 0, 0 >>>(i);
        else
            foo_kernel<<< 1, 1, 0, ls_stream[i] >>>(i);

    // synchronize the host and GPU
    hipDeviceSynchronize();

    // terminates all the created CUDA streams
    for (int i = 0; i < n_stream; i++)
        hipStreamDestroy(ls_stream[i]);
    delete [] ls_stream;

    return 0;
}