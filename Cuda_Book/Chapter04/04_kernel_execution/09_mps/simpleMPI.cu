#include "hip/hip_runtime.h"
#include <cstdio>
#include <omp.h>
#include <mpi.h>
#include <helper_timer.h>

using namespace std;

__global__ void vecAdd_kernel(float *c, const float* a, const float* b);
void init_buffer(float *data, const int size);

class Operator
{
private:
    int _index;
    hipStream_t stream;
    StopWatchInterface *p_timer;

    static void CUDART_CB Callback(hipStream_t stream, hipError_t status, void* userData);
    void print_time();

public:
    Operator() {
        hipStreamCreate(&stream);
        sdkCreateTimer(&p_timer);
    }

    ~Operator() {
        hipStreamDestroy(stream);
        sdkDeleteTimer(&p_timer);
    }

    void set_index(int idx) { _index = idx; }
    void async_operation(float *h_c, const float *h_a, const float *h_b,
                          float *d_c, float *d_a, float *d_b,
                          const int size, const int bufsize);
    
}; // Operator

void Operator::CUDART_CB Callback(hipStream_t stream, hipError_t status, void* userData) {
    Operator* this_ = (Operator*) userData;
    this_->print_time();
}

void Operator::print_time() {
    sdkStopTimer(&p_timer);    // end timer
    float elapsed_time_msed = sdkGetTimerValue(&p_timer);
    printf("stream %2d - elapsed %.3f ms \n", _index, elapsed_time_msed);
}

void Operator::async_operation(float *h_c, const float *h_a, const float *h_b,
                          float *d_c, float *d_a, float *d_b,
                          const int size, const int bufsize)
{
    // start timer
    sdkStartTimer(&p_timer);

    // copy host -> device
    hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice, stream);

    // launch cuda kernel
    dim3 dimBlock(256);
    dim3 dimGrid(size / dimBlock.x);
    vecAdd_kernel<<< dimGrid, dimBlock, 0, stream >>>(d_c, d_a, d_b);

    // copy device -> host
    hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost, stream);

    // register callback function
    hipStreamAddCallback(stream, Operator::Callback, this, 0);
}

int main(int argc, char *argv[])
{
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 24;
    int bufsize = size * sizeof(float);
    int num_operator = 4;

    if (argc != 1)
        num_operator = atoi(argv[1]);

    // initialize timer
    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    
    // set num_operator as the number of requested process
    int np, rank;
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &np);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    if (rank == 0)
        printf("Number of process: %d\n", np);

    bufsize /= np;
    size /= np;

    // allocate unified memory
    hipHostMalloc((void**)&h_a, bufsize);
    hipHostMalloc((void**)&h_b, bufsize);
    hipHostMalloc((void**)&h_c, bufsize);

    // initialize host values
    srand(2019);
    init_buffer(h_a, size);
    init_buffer(h_b, size);
    init_buffer(h_c, size);

    // allocate device memories
    hipMalloc((void**)&d_a, bufsize);
    hipMalloc((void**)&d_b, bufsize);
    hipMalloc((void**)&d_c, bufsize);

    printf("Number of operations: %d\n", num_operator);

    // create list of operation elements
    Operator *ls_operator = new Operator[num_operator];

    sdkStartTimer(&timer);
    
    // execute each operator collesponding data
    omp_set_num_threads(num_operator);
    #pragma omp parallel
    {
        int i = omp_get_thread_num();
        int offset = i * size / num_operator;
        printf("Launched GPU task (%d, %d)\n", rank, i);

        ls_operator[i].set_index(i);
        ls_operator[i].async_operation(&h_c[offset], &h_a[offset], &h_b[offset],
                                    &d_c[offset], &d_a[offset], &d_b[offset],
                                    size / num_operator, bufsize / num_operator);
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    // print out the result
    int print_idx = 256;
    printf("compared a sample result...\n");
    printf("host: %.6f, device: %.6f\n",  h_a[print_idx] + h_b[print_idx], h_c[print_idx]);

    // Compute and print the performance
    float elapsed_time_msed = sdkGetTimerValue(&timer);
    float bandwidth = 3 * bufsize * sizeof(float) / elapsed_time_msed / 1e6;
    printf("Time= %.3f msec, bandwidth= %f GB/s\n", elapsed_time_msed, bandwidth);

    sdkDeleteTimer(&timer);

    // terminate operators
    delete [] ls_operator;

    // terminate device memories
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // terminate host memories
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    MPI_Finalize();
    
    return 0;
}

void init_buffer(float *data, const int size)
{
    for (int i = 0; i < size; i++) 
        data[i] = rand() / (float)RAND_MAX;
}

__global__ void
vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < 500; i++)
        c[idx] = a[idx] + b[idx];
}