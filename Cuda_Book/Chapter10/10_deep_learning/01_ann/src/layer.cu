#include "hip/hip_runtime.h"
#include "layer.h"

#include <random>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cassert>
#include <math.h>
#include <algorithm>

#include <sstream>
#include <fstream>
#include <iostream>

using namespace cudl;

/****************************************************************
 * Layer definition                                             *
 ****************************************************************/
Layer::Layer()
{
	/* do nothing */
}

Layer::~Layer()
{
#if (DEBUG_FORWARD > 0 || DEBUG_BACKWARD > 0)
	std::cout << "Destroy Layer: " << name_ << std::endl;
#endif

	if (output_       != nullptr)  delete output_;
	if (grad_input_   != nullptr)  delete grad_input_;

	if (weights_      != nullptr)  delete weights_;
	if (biases_       != nullptr)  delete biases_;
	if (grad_weights_ != nullptr)  delete grad_weights_;
	if (grad_biases_  != nullptr)  delete grad_biases_;
}

void Layer::init_weight_bias(unsigned int seed)
{
	checkCudaErrors(hipDeviceSynchronize());

	if (weights_ == nullptr || biases_ == nullptr)
		return;

	// Create random network
	std::random_device rd;
	std::mt19937 gen(seed == 0 ? rd() : static_cast<unsigned int>(seed));

	// He uniform distribution
	float range = sqrt(6.f / input_->size());	// He's initialization
	std::uniform_real_distribution<> dis(-range, range);

	for (int i = 0; i < weights_->len(); i++)
		weights_->ptr()[i] = static_cast<float>(dis(gen));
	for (int i = 0; i < biases_->len(); i++)
		biases_->ptr()[i] = 0.f;

	// copy initialized value to the device
	weights_->to(DeviceType::cuda);
	biases_->to(DeviceType::cuda);

	std::cout << ".. initialized " << name_ << " layer .." << std::endl;
}

void Layer::update_weights_biases(float learning_rate)
{
	float eps = -1.f * learning_rate;
	if (weights_ != nullptr && grad_weights_ != nullptr)
	{
#if (DEBUG_UPDATE)
		weights_->print(name_ + "::weights (before update)", true);
		grad_weights_->print(name_ + "::gweights", true);
#endif // DEBUG_UPDATE

		// w = w + eps * dw
		checkCublasErrors(
			hipblasSaxpy(cuda_->cublas(),
				weights_->len(),
				&eps,
				grad_weights_->cuda(), 1,
				weights_->cuda(), 1));

#if (DEBUG_UPDATE)
		weights_->print(name_ + "weights (after update)", true);
		// getchar();
#endif // DEBUG_UPDATE
	}

	if (biases_ != nullptr && grad_biases_ != nullptr)
	{
#if (DEBUG_UPDATE)
		biases_->print(name_ + "biases (before update)", true);
		grad_biases_->print(name_ + "gbiases", true);
#endif // DEBUG_UPDATE

		// b = b + eps * db
		checkCublasErrors(
			hipblasSaxpy(cuda_->cublas(),
				biases_->len(),
				&eps,
				grad_biases_->cuda(), 1,
				biases_->cuda(), 1));

#if (DEBUG_UPDATE)
		biases_->print(name_ + "biases (after update)", true);
		// getchar();
#endif // DEBUG_UPDATE
	}
}

float Layer::get_loss(Blob<float> *target)
{
	assert("No Loss layer has no loss." && false);
	return EXIT_FAILURE;
}

int Layer::get_accuracy(Blob<float> *target)
{
	assert("No Loss layer cannot estimate accuracy." && false);
	return EXIT_FAILURE;
}

int Layer::load_parameter()
{
	std::stringstream filename_weights, filename_biases;

	// load weights and biases pretrained parameters
	filename_weights << name_ << ".bin";
	if (weights_->file_read(filename_weights.str()))
		return -1;

	filename_biases << name_ << ".bias.bin";
	if (biases_->file_read(filename_biases.str()))
		return -2;

	std::cout << ".. loaded " << name_ << " pretrain parameter.." << std::endl;

	return 0;
}

int Layer::save_parameter()
{
	std::stringstream filename_weights, filename_biases;

	std::cout << ".. saving " << name_ << " parameter ..";
	
	// Write weights file
	if (weights_)
	{
		filename_weights << name_ << ".bin";
		if (weights_->file_write(filename_weights.str()))
			return -1;
	}
	
	// Write bias file
	if (biases_)
	{
		filename_biases << name_ << ".bias.bin";
		if (biases_->file_write(filename_biases.str()))
			return -2;
	}

	std::cout << " done .." << std::endl;

	return 0;
}

/****************************************************************
 * Dense Layer                                                  *
 ****************************************************************/

Dense::Dense(std::string name, int output_size)
{
	name_ = name;
	output_size_ = output_size;
}

Dense::~Dense()
{
	if (d_one_vec != nullptr) 
		hipFree(d_one_vec);
}

__global__ void init_one_vec(float* d_one_vec, size_t length)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= length) return;

	d_one_vec[i] = 1.f;
}

Blob<float> *Dense::forward(Blob<float> *input)
{
	// initialize weights and biases
	if (weights_ == nullptr)
	{
		// setup parameter size information
		input_size_  = input->c() * input->h() * input->w();
		
		// initialize weight, bias, and output
		weights_ = new Blob<float>(1, 1, input_size_, output_size_);
		biases_  = new Blob<float>(1, 1, output_size_);

	}

	// initilaize input and output
	if (input_ == nullptr || batch_size_ != input->n())
	{
		input_ = input;
		batch_size_  = input->n();

		if (output_ == nullptr)
			output_  = new Blob<float>(batch_size_, output_size_);
		else
			output_->reset(batch_size_, output_size_);
		
		output_->tensor();

		if (d_one_vec != nullptr)
			hipFree(d_one_vec);
		checkCudaErrors(hipMalloc((void**)&d_one_vec, sizeof(float) * batch_size_));
		init_one_vec<<< (batch_size_+BLOCK_DIM_1D-1)/BLOCK_DIM_1D, BLOCK_DIM_1D >>>(d_one_vec, batch_size_);

		// initialize weights and biases
		if (load_pretrain_ && !freeze_)
		{
			if (load_parameter())
			{
				std::cout << "error occurred.." << std::endl;
				exit(-1);
			}
		}
		else if (!freeze_)
		{
			init_weight_bias();
		}
		else
		{
			/* do nothing */
		}
	}


	// output = weights^T * input (without biases)
	checkCublasErrors(
		hipblasSgemm(cuda_->cublas(),
			HIPBLAS_OP_T, HIPBLAS_OP_N, 
			output_size_, batch_size_, input_size_,
			&cuda_->one,  
			weights_->cuda(), input_size_, 
			input_->cuda(), input_size_,
			&cuda_->zero, 
			output_->cuda(),  output_size_));

	// output += biases * d_one_vec^T
	checkCublasErrors(hipblasSgemm(cuda_->cublas(),
					HIPBLAS_OP_N, HIPBLAS_OP_N, 
					output_size_, batch_size_, 1,
					&cuda_->one, 
					biases_->cuda(), output_size_, 
					d_one_vec, 1, 
					&cuda_->one, 
					output_->cuda(), output_size_));

#if (DEBUG_DENSE & 0x01)
	input_->print(  name_ + "::input",  true);
	weights_->print(name_ + "::weight", true);
	biases_->print( name_ + "::bias",   true);
	output_->print( name_ + "::output", true);
#endif // DEBUG_DENSE

	return output_;
}

Blob<float> *Dense::backward(Blob<float> *grad_output)
{
	if (grad_weights_ == nullptr)
	{
		grad_weights_ = new Blob<float>(weights_->shape());
		grad_biases_  = new Blob<float>(biases_->shape());
	}

	if (grad_input_ == nullptr || batch_size_ != grad_output->n())
	{
		grad_output_  = grad_output;

		if (grad_input_ == nullptr)
			grad_input_   = new Blob<float>(input_->shape());
		else
			grad_input_->reset(input_->shape());
	}

	// db = (dy) * d_one_vec
	hipblasSgemv(cuda_->cublas(),
			HIPBLAS_OP_N,
			output_size_, batch_size_,
			&cuda_->one,
			grad_output_->cuda(), output_size_,
			d_one_vec, 1,
			&cuda_->zero,
			grad_biases_->cuda(), 1);

	// dw = x * (dy)^T
	hipblasSgemm(cuda_->cublas(),
		HIPBLAS_OP_N, HIPBLAS_OP_T,
		input_size_, output_size_, batch_size_,
		&cuda_->one,
		input_->cuda(),        input_size_,
		grad_output_->cuda(),  output_size_,
		&cuda_->zero,
		grad_weights_->cuda(), input_size_);

	// dx = W * dy
	if (!gradient_stop_)
		hipblasSgemm(cuda_->cublas(),
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			input_size_, batch_size_, output_size_,
			&cuda_->one,
			weights_->cuda(),     input_size_,
			grad_output_->cuda(), output_size_,
			&cuda_->zero, 
			grad_input_->cuda(),  input_size_);

#if (DEBUG_DENSE & 0x02)
	std::cout << name_ << "[BACKWARD]" << std::endl;
	grad_output->print(  name_ + "::gradients", true, grad_output->n());
	grad_weights_->print(name_ + "::gfilter", true);
	grad_biases_->print( name_ + "::gbias", true);
	if (!gradient_stop_)
		grad_input_->print(  name_ + "::gdata", true);
#endif // DEBUG_DENSE

	return grad_input_;
}

/****************************************************************
 * Activation Layer                                             *
 ****************************************************************/

Activation::Activation(std::string name, hipdnnActivationMode_t mode, float coef)
{
	name_ = name;
	mode_ = mode;
	coef_ = coef;

	hipdnnCreateActivationDescriptor(&act_desc_);
	hipdnnSetActivationDescriptor(act_desc_, mode, HIPDNN_PROPAGATE_NAN, coef);
}

Activation::~Activation()
{
	hipdnnDestroyActivationDescriptor(act_desc_);
}

Blob<float> *Activation::forward(Blob<float> *input)
{
	if (input_ == nullptr || batch_size_ != input->n())
	{
		input_ = input;
		input_desc_ = input->tensor();
		batch_size_  = input->n();

		if (output_ == nullptr)
			output_ = new Blob<float>(input->shape());
		else
			output_->reset(input->shape());

		output_desc_ = output_->tensor();
	}

	hipdnnActivationForward(cuda_->cudnn(),
		act_desc_,
		&cuda_->one,
		input_desc_,
		input->cuda(),
		&cuda_->zero,
		output_desc_,
		output_->cuda());

	return output_;
}

Blob<float> *Activation::backward(Blob<float> *grad_output)
{
	if (grad_input_ == nullptr || batch_size_ != grad_output->n())
	{
		grad_output_ = grad_output;

		if (grad_input_ == nullptr)
			grad_input_ = new Blob<float>(input_->shape());
		else
			grad_input_->reset(input_->shape());		
	}

	hipdnnActivationBackward(cuda_->cudnn(),
		act_desc_,
		&cuda_->one, 
		output_desc_, output_->cuda(), 
		output_desc_, grad_output->cuda(), 
		input_desc_, input_->cuda(), 
		&cuda_->zero, 
		input_desc_, grad_input_->cuda());

	return grad_input_;
}

/****************************************************************
 * Softmax definition                                           *
 ****************************************************************/

Softmax::Softmax(std::string name)
{
	name_ = name;
}

Softmax::~Softmax()
{

}

Blob<float> *Softmax::forward(Blob<float> *input)
{
	if (input_ == nullptr || batch_size_ != input->n())
	{
		input_ = input;
		input_desc_ = input->tensor();
		batch_size_  = input->n();
		
		if (output_ == nullptr)
			output_ = new Blob<float>(input->shape());
		else
			output_->reset(input->shape());		

		output_desc_ = output_->tensor();
	}

#if (DEBUG_SOFTMAX & 0x01)
	std::cout << name_ << "[FORWARD]" << std::endl;
	input_->print(name_ + "::input", true, input->n());
#endif

	checkCudnnErrors(
		hipdnnSoftmaxForward(cuda_->cudnn(), HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
			&cuda_->one,  input_desc_,  input->cuda(),
			&cuda_->zero, output_desc_, output_->cuda()));

#if (DEBUG_SOFTMAX & 0x01)
	output_->print(name_ + "::output", true, input->n());
#endif

	return output_;
}

Blob<float> *Softmax::backward(Blob<float> *target)
{
	checkCudaErrors(hipDeviceSynchronize());

	if (grad_input_ == nullptr || batch_size_ != target->n())
	{
		if (grad_input_ == nullptr)
			grad_input_ = new Blob<float>(input_->shape());
		else
		 	grad_input_->reset(input_->shape());
	}

	// set grad_input_ as predict
	checkCudaErrors(hipMemcpyAsync(grad_input_->cuda(), 
		output_->cuda(), output_->buf_size(), 
		hipMemcpyDeviceToDevice));
	// set grad_input_ = predict - target	
	checkCublasErrors(
		hipblasSaxpy(cuda_->cublas(), target->len(),
			&cuda_->minus_one, target->cuda(), 1,
			grad_input_->cuda(), 1));

	// normalize the grad_output by the batch size
	int grad_output_size = target->n() * target->c() * target->h() * target->w();
	float scale = 1.f / static_cast<float>(target->n());
	checkCublasErrors(hipblasSscal(cuda_->cublas(), grad_output_size, &scale, grad_input_->cuda(), 1));

#if (DEBUG_SOFTMAX & 0x02)
	std::cout << name_ << "[BACKWARD]" << std::endl;
	input_->print( name_ + "::input", true);
	output_->print(name_ + "::predict", true);
	target->print( name_ + "::y", true, target->n());
	grad_input_->print(name_ + "::dx", true, target->n());
#endif

	return grad_input_;
}

float Softmax::get_loss(Blob<float> *target)
{
	return loss_.loss(output_, target);
}

int Softmax::get_accuracy(Blob<float> *target)
{
	int batch_size = output_->n();
	int output_size = output_->size();

	assert(batch_size == target->n());
	assert(output_size == target->size());

	float *h_output, *h_target;
	int idx_output, idx_target;
	int hit_count = 0;

	// get predicts and targets
	h_output = output_->to(host);
	h_target = target->to(host);

	// idx_output = idx_target = 0;
	for (int b = 0; b < batch_size; b++)
	{
		idx_output = 0;
		idx_target = 0;

		for (int i = 1; i < 10; i++)
		{
			if (h_output[b * output_size + i] > h_output[b * output_size + idx_output])
				idx_output = i;
			if (h_target[b * output_size + i] > h_target[b * output_size + idx_target])
				idx_target = i;
		}

		if (idx_output == idx_target)
			hit_count++;
	}

	return hit_count;
}

/****************************************************************
 * Layer definition                                             *
 ****************************************************************/

