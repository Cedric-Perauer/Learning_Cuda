#include "hip/hip_runtime.h"
#include "scan.h"

__global__ void
scan_v2_kernel(float *d_output, float *d_input, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    extern __shared__ float s_buffer[];
    s_buffer[threadIdx.x] = d_input[idx];
    s_buffer[threadIdx.x + BLOCK_DIM] = d_input[idx + BLOCK_DIM];

    int offset = 1;

    while (offset < length)
    {
        __syncthreads();

        int idx_a = offset * (2 * tid + 1) - 1;
        int idx_b = offset * (2 * tid + 2) - 1;

        if (idx_a >= 0 && idx_b < 2 * BLOCK_DIM)
        {
#if (DEBUG_INDEX > 0)
            printf("[ %d, %d ]\t", idx_a, idx_b);
#endif
            s_buffer[idx_b] += s_buffer[idx_a];
        }

        offset <<= 1;
#if (DEBUG_INDEX > 0)
        if (tid == 0)   printf("\n--------------------------------\n");
#endif  
    }

    offset >>= 1;
    while (offset > 0)
    {
        __syncthreads();

        int idx_a = offset * (2 * tid + 2) - 1;
        int idx_b = offset * (2 * tid + 3) - 1;

        if (idx_a >= 0 && idx_b < 2 * BLOCK_DIM)
        {
#if (DEBUG_INDEX > 0)
            printf("[ %d, %d ]\t", idx_a, idx_b);
#endif
            s_buffer[idx_b] += s_buffer[idx_a];
        }

        offset >>= 1;
#if (DEBUG_INDEX > 0)
        if (tid == 0)   printf("\n--------------------------------\n");
#endif  
    }
    __syncthreads();

    d_output[idx] = s_buffer[tid];
    d_output[idx + BLOCK_DIM] = s_buffer[tid + BLOCK_DIM];
}

void scan_v2(float *d_output, float *d_input, int length)
{
    dim3 dimBlock(BLOCK_DIM);
    dim3 dimGrid((length + (2 * BLOCK_DIM) - 1) / (2 * BLOCK_DIM));
    scan_v2_kernel<<<dimGrid, dimBlock, sizeof(float) * BLOCK_DIM * 2>>>(d_output, d_input, length);
    hipDeviceSynchronize();
}