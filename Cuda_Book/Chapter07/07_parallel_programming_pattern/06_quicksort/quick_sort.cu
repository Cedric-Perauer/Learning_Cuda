#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#define MAX_DEPTH       24
#define INSERTION_SORT  32

// use selection sort when data reaches the max depth level
__device__ void selection_sort( unsigned int *data, int left, int right )
{
  for( int i = left ; i <= right ; ++i )
  {
    unsigned min_val = data[i];
    int min_idx = i;

    // Find the smallest value in the range [left, right].
    for( int j = i+1 ; j <= right ; ++j )
    {
      unsigned val_j = data[j];
      if( val_j < min_val )
      {
        min_idx = j;
        min_val = val_j;
      }
    }

    // Swap the values.
    if( i != min_idx )
    {
      data[min_idx] = data[i];
      data[i] = min_val;
    }
  }
}
// Quicksort algorithm making use of Dynamic Parallelism sorting requirsively till max depth is recched
__global__ void cdp_simple_quicksort( unsigned int *data, int left, int right, int depth )
{
  if( depth >= MAX_DEPTH || right-left <= INSERTION_SORT )
  {
    selection_sort( data, left, right );
    return;
  }

  unsigned int *lptr = data+left;
  unsigned int *rptr = data+right;
  unsigned int  pivot = data[(left+right)/2];

  // Do the partitioning.
  while(lptr <= rptr)
  {
    // Find the next left- and right-hand values to swap
    unsigned int lval = *lptr; 
    unsigned int rval = *rptr;

    // Move the left pointer as long as the pointed element is smaller than the pivot.
    while( lval < pivot )
    {
      lptr++;
      lval = *lptr;
    }

    // Move the right pointer as long as the pointed element is larger than the pivot.
    while( rval > pivot )
    {
      rptr--;
      rval = *rptr;
    }

    // If the swap points are valid, do the swap!
    if(lptr <= rptr)
    {
      *lptr++ = rval;
      *rptr-- = lval;
    }
  }

  // Now the recursive part
  int nright = rptr - data;
  int nleft  = lptr - data;

  // Launch a new block to sort the left part.
  if(left < (rptr-data)) 
  {
    hipStream_t s;
    hipStreamCreateWithFlags( &s, hipStreamNonBlocking );
    cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
    hipStreamDestroy( s );
  }

  // Launch a new block to sort the right part.
  if((lptr-data) < right) 
  {
    hipStream_t s1;
    hipStreamCreateWithFlags( &s1, hipStreamNonBlocking );
    cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
    hipStreamDestroy( s1 );
  }
}

// Call the quicksort kernel from the host.
void run_qsort(unsigned int *data, unsigned int nitems)
{
  // Prepare CDP for the max depth 'MAX_DEPTH'.
  checkCudaErrors( hipDeviceSetLimit( cudaLimitDevRuntimeSyncDepth, MAX_DEPTH ) );

  // Launch on device
  int left = 0;
  int right = nitems-1;
  std::cout << "Launching kernel on the GPU" << std::endl;
  cdp_simple_quicksort<<< 1, 1 >>>(data, left, right, 0);
  checkCudaErrors(hipDeviceSynchronize());
}

// Initialize data on the host.
void initialize_data(unsigned int *dst, unsigned int nitems)
{
  // Fixed seed for illustration
  srand(2047);

  // Fill dst with random values
  for (unsigned i = 0 ; i < nitems ; i++)
    dst[i] = rand() % nitems ;
}

// Verify the results.
void check_results( int n, unsigned int *results_d )
{
  unsigned int *results_h = new unsigned[n];
  checkCudaErrors( hipMemcpy( results_h, results_d, n*sizeof(unsigned), hipMemcpyDeviceToHost ));
  for( int i = 1 ; i < n ; ++i )
    if( results_h[i-1] > results_h[i] )
    {
      std::cout << "Invalid item[" << i-1 << "]: " << results_h[i-1] << " greater than " << results_h[i] << std::endl;
      exit(EXIT_FAILURE);
    }
  std::cout << "OK" << std::endl;
  delete[] results_h;
}

int main(int argc, char **argv)
{
  int num_items = 2048;

  // Get device properties
  hipDeviceProp_t properties;
  checkCudaErrors( hipGetDeviceProperties( &properties, 0 ) );
  if(!(( properties.major >= 3)|| ( properties.major == 3 && properties.minor >= 5 ) ))
  {
    std::cerr << "cdpSimpleQuicksort requires GPU devices with compute SM 3.5 or higher.  Exiting..." << std::endl;
    exit(0);
  }

  // Create input data
  unsigned int *h_data = 0;
  unsigned int *d_data = 0;

  // Allocate CPU memory and initialize data.
  h_data =(unsigned int *)malloc( num_items*sizeof(unsigned int));
  initialize_data(h_data, num_items);
  
  // Allocate GPU memory.
  checkCudaErrors(hipMalloc((void **)&d_data, num_items * sizeof(unsigned int)));
  checkCudaErrors(hipMemcpy(d_data, h_data, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));

  // Execute
  std::cout << "Running quicksort on " << num_items << " elements" << std::endl;
  run_qsort(d_data, num_items);
  
  // Check result
  std::cout << "Validating results: ";
  check_results(num_items, d_data);

  free(h_data);
  checkCudaErrors( hipFree(d_data));
}

