#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "../03_scan/utils.h"
#include "../03_scan/scan_v2.cu"

#define FLT_ZERO 0.f
#define GRID_DIM    1       // this implementation covers only 1 thread block's operation

// #define BLOCK_DIM 16

void generate_data(float *ptr, int length);

// predicate
// mark elements which will be scattered
__global__ void
predicate_kernel(float *d_predicates, float *d_input, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= length) return;

    d_predicates[idx] = d_input[idx] > FLT_ZERO;
}

// scan
/* We will use the previous implementation (#include "../03_scan/scan_v2.cu") */

// address and gather
__global__ void
pack_kernel(float *d_output, float *d_input, float *d_predicates, float *d_scanned, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= length) return;

    if (d_predicates[idx] != 0.f)
    {
        // address
        int address = d_scanned[idx] - 1;

        // gather
        d_output[address] = d_input[idx];
    }
}

__global__ void
split_kernel(float *d_output, float *d_input, float *d_predicates, float *d_scanned, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= length) return;

    if (d_predicates[idx] != 0.f)
    {
        // address
        int address = d_scanned[idx] - 1;

        // split
        d_output[idx] = d_input[address];
    }
}

// pack_host : evaluation purpose
void pack_host(float *h_output, float *h_input, int length)
{
    int idx_output = 0;
    for (int i = 0; i < length; i++)
    {
        if (h_input[i] > FLT_ZERO)
        {
            h_output[idx_output] = h_input[i];
            idx_output++;
        }
    }
}

// split_host: pseudo implementation for evaluation purpose
void split_host(float *h_output, float *h_input, int length)
{
    for (int i = 0; i < length; i++)
    {
        if (h_input[i] >= 0.f)
            h_output[i] = h_input[i];
        else
            h_output[i] = 0.f;
    }
}

int main()
{
    float *h_input, *h_output_host, *h_output_gpu;
    float *d_input, *d_output;
    float *d_predicates, *d_scanned; // for temporarly purpose operation
    float length = BLOCK_DIM;

    srand(2019);

    // allocate host memory
    h_input = (float *)malloc(sizeof(float) * length);
    h_output_host = (float *)malloc(sizeof(float) * length);
    h_output_gpu = (float *)malloc(sizeof(float) * length);

    // allocate device memory
    hipMalloc((void**)&d_input, sizeof(float) * length);
    hipMalloc((void**)&d_output, sizeof(float) * length);
    hipMalloc((void**)&d_predicates, sizeof(float) * length);
    hipMalloc((void**)&d_scanned, sizeof(float) * length);

    // generate input data
    generate_data(h_input, length);
    hipMemcpy(d_input, h_input, sizeof(float) * length, hipMemcpyHostToDevice);

    print_val(h_input, DEBUG_OUTPUT_NUM, "input    ::");

    hipProfilerStart();
    /********************************
     * Pack                         *
     ********************************/
    // predicates
    predicate_kernel<<< GRID_DIM, BLOCK_DIM >>>(d_predicates, d_input, length);

    // scan
    scan_v2(d_scanned, d_predicates, length);

    // addressing & gather (pack)
    pack_kernel<<< GRID_DIM, BLOCK_DIM >>>(d_output, d_input, d_predicates, d_scanned, length);
    hipDeviceSynchronize();

    // validation the result (compack)
    hipMemcpy(h_output_gpu, d_output, sizeof(float) * length, hipMemcpyDeviceToHost);
    pack_host(h_output_host, h_input, length);

    print_val(h_output_host, DEBUG_OUTPUT_NUM, "pack[cpu]::");
    print_val(h_output_gpu, DEBUG_OUTPUT_NUM, "pack[gpu]::");

    if (validation(h_output_host, h_output_gpu, length))
        printf("SUCCESS!!\n");
    else
        printf("Something wrong..\n");

    /********************************
     * Split                        *
     ********************************/
    hipMemcpy(d_input, d_output, sizeof(float) * length, hipMemcpyDeviceToDevice);
    hipMemset(d_output, 0, sizeof(float) * length);
    split_kernel<<<GRID_DIM, BLOCK_DIM>>>(d_output, d_input, d_predicates, d_scanned, length);
    hipDeviceSynchronize();
    hipProfilerStop();

    // validation the result (split)
    hipMemcpy(h_output_gpu,  d_output, sizeof(float) * length, hipMemcpyDeviceToHost);
    split_host(h_output_host, h_input, length); // notice: we just generate desired output for the evaluation purpose

    print_val(h_output_gpu, DEBUG_OUTPUT_NUM, "split[gpu]");
    if (validation(h_output_host, h_output_gpu, length))
        printf("SUCCESS!!\n");
    else
        printf("Something wrong..\n");

    // finalize
    hipFree(d_predicates);
    hipFree(d_scanned);
    hipFree(d_input);
    hipFree(d_output);
    free(h_output_gpu);
    free(h_output_host);
    free(h_input);

    return 0;
}


