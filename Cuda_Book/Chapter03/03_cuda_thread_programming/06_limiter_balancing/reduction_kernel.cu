#include "hip/hip_runtime.h"
#include <stdio.h>
#include "reduction.h"

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/
__global__ void
reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    // cumulates input with grid-stride loop and save to share memory
    float input = 0.f;
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x)
        input += g_in[i];
    s_data[threadIdx.x] = input;

    __syncthreads();

    // do reduction
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride) 
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];

        __syncthreads();
    }

    if (threadIdx.x == 0) {
        g_out[blockIdx.x] = s_data[0];
    }
}

int reduction(float *g_outPtr, float *g_inPtr, int size, int n_threads)
{
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1) / n_threads);

    reduction_kernel<<<n_blocks, n_threads, n_threads * sizeof(float), 0>>>(g_outPtr, g_inPtr, size);
    reduction_kernel<<<1, n_threads, n_threads * sizeof(float), 0>>>(g_outPtr, g_inPtr, n_blocks);

    return 1;
}