#include "hip/hip_runtime.h"
#include <stdio.h>
#include "reduction.h"

#define NUM_LOAD 4

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/**
    Two warp level primitives are used here for this example
    https://devblogs.nvidia.com/faster-parallel-reductions-kepler/
    https://devblogs.nvidia.com/using-cuda-warp-level-primitives/
 */

__inline__ __device__ float warp_reduce_sum(float val)
{
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        unsigned int mask = __activemask();
        val += __shfl_down_sync(mask, val, offset);
    }
    return val;
}

__inline__ __device__ float block_reduce_sum(float val)
{
    static __shared__ float shared[32]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    // Each warp performs partial reduction
    val = warp_reduce_sum(val); 

    if (lane == 0)
        shared[wid] = val; // Write reduced value to shared memory

    __syncthreads(); // Wait for all partial reductions

    //read from shared memory only if that warp existed
    if (wid == 0) {
        val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
        val = warp_reduce_sum(val); //Final reduce within first warp
    }

    return val;
}

// cuda thread synchronization
__global__ void
reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    // cumulates input with grid-stride loop and save to share memory
    float sum[NUM_LOAD] = { 0.f };
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x * NUM_LOAD)
    {
        for (int step = 0; step < NUM_LOAD; step++)
            sum[step] += (i + step * blockDim.x * gridDim.x < size) ? g_in[i + step * blockDim.x * gridDim.x] : 0.f;
    }
    for (int i = 1; i < NUM_LOAD; i++)
        sum[0] += sum[i];
    // warp synchronous reduction
    sum[0] = block_reduce_sum(sum[0]);

    if (threadIdx.x == 0)
        g_out[blockIdx.x] = sum[0];
}

void reduction(float *g_outPtr, float *g_inPtr, int size, int n_threads)
{
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1) / n_threads);

    reduction_kernel<<<n_blocks, n_threads>>>(g_outPtr, g_inPtr, size);
    reduction_kernel<<< 1, n_threads, n_threads * sizeof(float), 0 >>>(g_outPtr, g_inPtr, n_blocks);
}
