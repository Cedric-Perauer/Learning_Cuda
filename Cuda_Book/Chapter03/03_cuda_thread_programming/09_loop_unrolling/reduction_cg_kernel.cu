#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include "reduction.h"

using namespace cooperative_groups;

#define NUM_LOAD 4

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

template <typename group_t>
__inline__ __device__ float warp_reduce_sum(group_t group, float val)
{
    #pragma unroll 5
    for (int offset = group.size() / 2; offset > 0; offset >>= 1)
        val += group.shfl_down(val, offset);
    return val;
}

__inline__ __device__ float block_reduce_sum(thread_block block, float val)
{
    __shared__ float shared[32]; // Shared mem for 32 partial sums
    // int warp_idx = threadIdx.x / warpSize;
    int warp_idx = block.thread_index().x / warpSize;

    // partial reduciton at tile<32> size
    thread_block_tile<32> tile32 = tiled_partition<32>(block);
    val = warp_reduce_sum(tile32, val);

    // write reduced value to shared memory
    if (tile32.thread_rank() == 0)
        shared[warp_idx] = val; 

    block.sync(); // Wait for all partial reductions

    //read from shared memory only if that warp existed
    if (warp_idx == 0) {
        val = (block.thread_index().x < blockDim.x / warpSize) ? shared[tile32.thread_rank()] : 0;
        val = warp_reduce_sum(tile32, val); //Final reduce within first warp
    }

    return val;
}

// large vector reduction
__global__ void
reduction_kernel(float* g_out, float* g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    thread_block block = this_thread_block();

    // cumulates input with grid-stride loop and save to share memory
    float sum[NUM_LOAD] = { 0.f };
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x * NUM_LOAD)
    {
        for (int step = 0; step < NUM_LOAD; step++)
            sum[step] += (i + step * blockDim.x * gridDim.x < size) ? g_in[i + step * blockDim.x * gridDim.x] : 0.f;
    }
    for (int i = 1; i < NUM_LOAD; i++)
        sum[0] += sum[i];
        
    // warp synchronous reduction
    sum[0] = block_reduce_sum(block, sum[0]);

    if (block.thread_index().x == 0)
        g_out[block.group_index().x] = sum[0];
}

void reduction(float *g_outPtr, float *g_inPtr, int size, int n_threads)
{   
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1) / n_threads);

    reduction_kernel<<< n_blocks, n_threads>>>(g_outPtr, g_inPtr, size);
    reduction_kernel<<< 1, n_threads >>>(g_outPtr, g_inPtr, n_blocks);
}
