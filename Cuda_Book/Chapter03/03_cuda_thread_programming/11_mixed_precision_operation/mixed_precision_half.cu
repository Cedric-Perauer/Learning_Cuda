#include "hip/hip_runtime.h"

// https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH____HALF__ARITHMETIC.html
// FP16 types and intrinsics
#include <hip/hip_fp16.h>
#include <stdlib.h>
#include <helper_timer.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include "util.cuh"
 
using namespace cooperative_groups;

// FMA numerical arithmetic function in GPU @FP16
// y = x * y + z
// in this kernel, assuming we have transposed matrix y
__global__ void hfma_kernel(half *d_x, half *d_y, float *d_z, int size)
{
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    half2 *dual_x = reinterpret_cast<half2*>(d_x);
    half2 *dual_y = reinterpret_cast<half2*>(d_y);
    float2 *dual_z = reinterpret_cast<float2*>(d_z);

    extern __shared__ float2 s_data[];

#if __CUDA_ARCH__ >= 530
    for (int i = idx_x; i < size; i+=stride) {
        dual_z[i] = __half22float2(__hmul2(dual_y[i], dual_x[i]));
    }
#else
    for (int i = idx_x; i < size; i+=stride) {
        dual_z[i] = __half22float2(dual_x[i]) * __half22float2(dual_y[i]);
    }
#endif
}

void fhma_host(half *h_x, half *h_y, float *h_z, int size)
{
    #pragma omp parallel
    {
    #pragma omp for
        for (int i = 0; i < size; i++)
            h_z[i] = __half2float(h_y[i]) * __half2float(h_x[i]);
    }
}

int main()
{
    CBuffer<half> X, Y;
    CBuffer<float> Z;
    int size = 1 << 26;

    srand(2019);

    // initialize host buffers
    X.init(size, true);
    Y.init(size, true);
    Z.init(size, true);

    // initalize gpu buffers
    X.cuda();
    Y.cuda();
    Z.cuda();

    // getting number of blocks for stride-loop
    int n_threads = 256;
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, hfma_kernel, n_threads, n_threads*sizeof(float2));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size/2 + n_threads - 1) / n_threads);

    // initialize timer
    StopWatchInterface *timer;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    hfma_kernel<<< n_blocks, n_threads, n_threads * sizeof(float2) >>>(X.d_ptr_, Y.d_ptr_, Z.d_ptr_, size/2);

    hipDeviceSynchronize();
    sdkStopTimer(&timer);

    float elapsedTimeMs = sdkGetTimerValue(&timer);
    double ops = (float)size / elapsedTimeMs * 1e-6;
    printf("FMA, FLOPS = %.3f GFlops, Operation Time= %.3f msec\n", ops, elapsedTimeMs);

    fhma_host(X.h_ptr_, Y.h_ptr_, Z.h_ptr_, size);

    int diff_count = Z.diff_count();
    (diff_count == 0) ? printf("Success!!\n") : printf("Counted diff!! (%d times)\n", diff_count);

    // cleanup
    sdkDeleteTimer(&timer);
    
    return 0;
}