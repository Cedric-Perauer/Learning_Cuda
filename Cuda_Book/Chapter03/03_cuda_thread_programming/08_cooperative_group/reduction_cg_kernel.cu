#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include "reduction.h"

using namespace cooperative_groups;

#define NUM_LOAD 4

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

// cuda thread synchronization
__global__ void
reduction_kernel(float* g_out, float* g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    thread_block block = this_thread_block();

    extern __shared__ float s_data[];

    // cumulates input with grid-stride loop and save to share memory
    float input[NUM_LOAD] = {0.f};
    for (int i = idx_x; i < size; i += block.group_dim().x * gridDim.x * NUM_LOAD)
    {
        for (int step = 0; step < NUM_LOAD; step++)
            input[step] += (i + step * block.group_dim().x * gridDim.x < size) ? g_in[i + step * block.group_dim().x * gridDim.x] : 0.f;
    }
    for (int i = 1; i < NUM_LOAD; i++)
        input[0] += input[i];
    s_data[threadIdx.x] = input[0];

    block.sync();

    // do reduction
    for (unsigned int stride = block.group_dim().x / 2; stride > 0; stride >>= 1) {
        // scheduled threads reduce for every iteration
        // and will be smaller than a warp size (32) eventually.
        if (block.thread_index().x < stride) { 
            s_data[block.thread_index().x] += s_data[block.thread_index().x + stride];

            // __syncthreads(); // Step 3: Error
            // block.sync();    // Step 4: Benefit of cooperative group, performance may drop but provides programming flexibility
        }
        // __syncthreads(); // Step 1: Original
        block.sync();       // Step 2: Equivalent operation
    }

    if (block.thread_index().x == 0) {
        g_out[block.group_index().x] = s_data[0];
    }
}

void reduction(float *g_outPtr, float *g_inPtr, int size, int n_threads)
{
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1) / n_threads);

    reduction_kernel<<< n_blocks, n_threads, n_threads * sizeof(float), 0 >>>(g_outPtr, g_inPtr, size);
    reduction_kernel<<< 1, n_threads, n_threads * sizeof(float), 0 >>>(g_outPtr, g_inPtr, n_blocks);
}
