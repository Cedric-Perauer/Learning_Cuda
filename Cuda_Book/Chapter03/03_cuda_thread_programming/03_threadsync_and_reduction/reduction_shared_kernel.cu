#include "hip/hip_runtime.h"
#include <stdio.h>
#include "reduction.h"

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

// cuda thread synchronization
__global__ void
reduction_kernel(float* d_out, float* d_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    s_data[threadIdx.x] = (idx_x < size) ? d_in[idx_x] : 0.f;

    __syncthreads();

    // do reduction
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        // thread synchronous reduction
        if ( (idx_x % (stride * 2)) == 0 )
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];

        __syncthreads();
    }

    if (threadIdx.x == 0)
        d_out[blockIdx.x] = s_data[0];
}

void reduction(float *d_out, float *d_in, int n_threads, int size)
{   
    hipMemcpy(d_out, d_in, size * sizeof(float), hipMemcpyDeviceToDevice);
    while(size > 1)
    {
        int n_blocks = (size + n_threads - 1) / n_threads;
        reduction_kernel<<< n_blocks, n_threads, n_threads * sizeof(float), 0 >>>(d_out, d_out, size);
        size = n_blocks;
    } 
}