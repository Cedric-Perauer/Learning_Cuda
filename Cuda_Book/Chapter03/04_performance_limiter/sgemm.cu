
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_DIM 16

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on GPU
//! C = alpha * A * B + beta * C
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param C          matrix C as provided to device
//! @param N          height of matrix A and matrix C
//! @param M          width of matrix B and matrix C
//! @param K          width of matrix A and height of matrix C
//! @param alpha      scala value for matrix multiplication
//! @param beta       scala value for matrix summation with C
////////////////////////////////////////////////////////////////////////////////
__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, 
                            int N, int M, int K, 
                            float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;
    for (int i = 0; i < K; ++i) {
        sum += A[row * K + i] * B[i * K + col];
    }
    C[row * M + col] = alpha * sum + beta * C[row * M + col];
}

void sgemm_gpu(const float *A, const float *B, float *C, 
            int N, int M, int K, 
            float alpha, float beta)
{
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid(M / dimBlock.x, N / dimBlock.y);
    sgemm_gpu_kernel<<< dimGrid, dimBlock >>>(A, B, C, N, M, K, alpha, beta);
}

void random_init(float *data, int size)
{
    for (int i = 0; i < size; ++i) {
        data[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }
}


int main()
{
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    int N, M, K;
    float alpha = 2.f;
    float beta = 1.f;
    N = M = K = 2048;

    // allocation of linear memory space
    A = (float *)malloc(N * K * sizeof(float));
    B = (float *)malloc(K * M * sizeof(float));
    C = (float *)malloc(N * M * sizeof(float));

    // allocation of gpu linear memory space
    hipMalloc((void **)&d_A, N * K * sizeof(float));
    hipMalloc((void **)&d_B, K * M * sizeof(float));
    hipMalloc((void **)&d_C, N * M * sizeof(float));

    // initialize randomized values for memory space
    random_init(A, N * K);
    random_init(B, K * M);
    random_init(C, N * M);

    // copy initial value for gpu memory
    hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, A, K * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, A, N * M * sizeof(float), hipMemcpyHostToDevice);

    // do operation
    sgemm_gpu(d_A, d_B, d_C, N, M, K, alpha, beta);

    // terminates allocated gpu memory space
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // terminates allocated memory space
    free(A);
    free(B);
    free(C);

    return 0;
}