
#include <hip/hip_runtime.h>


__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < len)
    out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  int inputLength  = 1<<28;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;


  /*hostInput1 = (float*) malloc (sizeof(float) * inputLength);
  hostInput2 = (float*) malloc (sizeof(float) * inputLength);
  hostOutput = (float*) malloc (sizeof(float) * inputLength);a*/
  hipHostMalloc(&hostInput1, inputLength*sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&hostInput2, inputLength*sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&hostOutput, inputLength*sizeof(float), hipHostMallocDefault);

  for(int i=0;i<inputLength;i++) {
    hostInput1[i] = i%1024;
    hostInput2[i] = i%1024;
  }
  
  hipStream_t stream[4];
  float *d_A[4], *d_B[4], *d_C[4];
  int i, k, Seglen = 16384;
  int Gridlen = (Seglen - 1) / 256 + 1;
  
  for (i = 0; i < 4; i++) {
    hipStreamCreateWithFlags(&stream[i],hipStreamNonBlocking);
    hipMalloc((void **)&d_A[i], Seglen * sizeof(float));
    hipMalloc((void **)&d_B[i], Seglen * sizeof(float));
    hipMalloc((void **)&d_C[i], Seglen * sizeof(float));
  }

  for (i = 0; i < inputLength; i += Seglen * 4) {
    for (k = 0; k < 4; k++) {
     
      hipMemcpyAsync(d_A[k], hostInput1 + i + k * Seglen,
                      Seglen * sizeof(float), hipMemcpyHostToDevice,
                      stream[k]);
      hipMemcpyAsync(d_B[k], hostInput2 + i + k * Seglen,
                      Seglen * sizeof(float), hipMemcpyHostToDevice,
                      stream[k]);
      vecAdd<<<Gridlen, 256, 0, stream[k]>>>(d_A[k], d_B[k], d_C[k],
                                             Seglen);
    }
    hipStreamSynchronize(stream[0]);
    hipStreamSynchronize(stream[1]);
    hipStreamSynchronize(stream[2]);
    for (k = 0; k < 4; k++) {
      hipMemcpyAsync(hostOutput + i + k * Seglen, d_C[k],
                      Seglen * sizeof(float), hipMemcpyDeviceToHost,
                      stream[k]);
    }
  }
  hipDeviceSynchronize();


  hipHostFree(hostInput1);
  hipHostFree(hostInput2);
  hipHostFree(hostOutput);
  
  for (k = 0; k < 3; k++) {
    hipFree(d_A[k]);
    hipFree(d_B[k]);
    hipFree(d_C[k]);
  }

  return 0;
}

