#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <hiprand.h>
#include "hip/hip_runtime_api.h"
#include "helper.cuh"

typedef hipfftReal    Real;
typedef hipfftComplex Complex;

int main(int argc, char *argv[])
{
    long long sample_size = 1 << 20;      // 1,048,576
    const int batch_size = 1 << 9;  // 512
    int n_gpu = 2;

    hipfftHandle cufft_plan;
    Complex *d_input;
    Complex *h_input, *h_output;

    float forward_time_ms, inverse_time_ms;

    // create cuda event to measure the performance
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_input = (Complex*) new Complex[sample_size * batch_size];
    h_output= (Complex*) new Complex[sample_size * batch_size];

    // create hiprand generator & set random seed
    hiprandGenerator_t curand_gen;
    hipMalloc((void**)&d_input, sizeof(Complex) * sample_size * batch_size);
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 2019UL);
    op::hiprand(curand_gen, d_input, sample_size * batch_size);
    hipMemcpy(h_input, d_input, sizeof(Complex) * sample_size * batch_size, hipMemcpyDeviceToHost);
    hipFree(d_input);

    // 1. create cufft empty plan
    hipfftCreate(&cufft_plan);

    // 2. set multi-gpu
    int *devices = (int*) new int[n_gpu];
    for (int i = 0; i < n_gpu; i++)
        devices[i] = i;
    hipfftXtSetGPUs(cufft_plan, n_gpu, devices);

    // 3. create teh plan
    size_t *workSize = (size_t*) new size_t[n_gpu];
    hipfftXtMakePlanMany(cufft_plan, 1, &sample_size,
                        nullptr, 1, 1, HIP_C_32F,
                        nullptr, 1, 1, HIP_C_32F,
                        batch_size, workSize, HIP_C_32F);

    // 4. allocate multi-gpu memory space and copy data from the host
    hipLibXtDesc *d_sample;
    checkCudaErrors(hipfftXtMalloc(cufft_plan, &d_sample, HIPFFT_XT_FORMAT_INPLACE));
    checkCudaErrors(hipfftXtMemcpy(cufft_plan, d_sample, h_input, HIPFFT_COPY_HOST_TO_DEVICE));

    // 5. executes FFT processes
    hipEventRecord(start);
    checkCudaErrors(hipfftXtExecDescriptor(cufft_plan, d_sample, d_sample, HIPFFT_FORWARD));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    checkCudaErrors(hipEventElapsedTime(&forward_time_ms, start, stop));

    // 6. executes FFT processes (inverse transformation)
    hipEventRecord(start);
    hipfftXtExecDescriptor(cufft_plan, d_sample, d_sample, HIPFFT_BACKWARD);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&inverse_time_ms, start, stop);

    // 7. copy the result to the host
    hipfftXtMemcpy(cufft_plan, h_output, d_sample, HIPFFT_COPY_DEVICE_TO_HOST);

    // print elapsed time
    std::cout << "FFT operation time for " << sample_size << " elements with " << batch_size << " batch.." << std::endl;
    std::cout << "Forward (ms): " << forward_time_ms << std::endl;
    std::cout << "Inverse (ms): " << inverse_time_ms << std::endl;

    // terminates used resources
    hiprandDestroyGenerator(curand_gen);

    // deletes CUFFT plan_forward
    hipfftDestroy(cufft_plan);

    // terminates memories
    hipfftXtFree(d_sample);
    
    // delete cuda events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    delete [] h_input;
    delete [] h_output;

    return 0;
}
