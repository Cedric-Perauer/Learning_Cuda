
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h> 

__global__ void print_from_gpu(void) {
	printf("Hello World! from thread [%d,%d] \
		From device\n", threadIdx.x,blockIdx.x); 
}

int main(void) { 
	printf("Hello World from host!\n"); 
	print_from_gpu<<<1,1>>>();
	hipDeviceSynchronize();
return 0; 
}

